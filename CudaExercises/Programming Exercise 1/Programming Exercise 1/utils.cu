﻿#include "hip/hip_runtime.h"
#include ""
#include "utils.cuh"
#include <iostream>
#include <random>


using namespace std;

void printCudaDeviceProperties(hipDeviceProp_t  &deviceProp) {
	cout << "===========================================================" << endl;
	cout << "Device " << deviceProp.name << " Properties:" << endl;
	cout << "Clock Rate (KHz) " << deviceProp.clockRate << endl;
	cout << "Memory Clock Rate (KHz) " << deviceProp.memoryClockRate << endl;
	cout << "Total Global Memory (MB):\t" << (deviceProp.totalGlobalMem / MB) << endl;
	cout << "Shared Memory / Block (KB):\t" << (deviceProp.sharedMemPerBlock / KB) << endl;
	// Warp: groups of threads that executes the same instruction
	cout << "Warp Size:\t" << deviceProp.warpSize << endl;
	// Pitch: Padded size of each row in an array?
	cout << "Pitch (MB):\t" << (deviceProp.memPitch / MB) << endl;
	cout << "Max Threads / Block:\t" << deviceProp.maxThreadsPerBlock << endl;
	cout << "Max Dimension Size of Block :\t" << deviceProp.maxThreadsDim[0] << ", " << deviceProp.maxThreadsDim[1] << ", " << deviceProp.maxThreadsDim[2] << endl;
	cout << "Max Dimension Size of Grid :\t" << deviceProp.maxGridSize[0] << ", " << deviceProp.maxGridSize[1] << ", " << deviceProp.maxGridSize[2] << endl;
	cout << "Multiprocess Count:\t" << deviceProp.multiProcessorCount << endl;
	cout << "Max Blocks / Multiprocessor:\t" << deviceProp.maxBlocksPerMultiProcessor << endl;
	cout << "Concurrent Kernels:\t" << deviceProp.concurrentKernels << endl;
	cout << "Max Threads / Multiprocessor:\t" << deviceProp.maxThreadsPerMultiProcessor << endl;
	cout << "Shared Memory (KB) / Multiprocessor:\t" << (deviceProp.sharedMemPerMultiprocessor / KB) << endl;
	cout << "===========================================================" << endl;
}

float* generateRandomMatrix(int size, pair<int, int> range) {
	random_device rd;
	mt19937 gen(rd());
	uniform_real_distribution<float> dis(range.first, range.second);
	
	float* matrix = new float [size*size];

	for (int i = 0;i < size;i++) {
		for (int j = 0;j < size;j++) {
			matrix[i*size+j] = dis(gen);
		}
	}
	return matrix;
}

void printMatrix(float* matrix, int size) {
	for (int i = 0;i < size;i++) {
		for (int j = 0;j < size;j++) {
			printf("%08.4f\t", matrix[i*size+j]);
		}
		cout << endl;
	}
}

void printVectorizedMatrix(float* matrix, pair<int, int> dim) {
	for (int i = 0;i < dim.first;i++) {
		for (int j = 0;j < dim.second;j++) {
			printf("%08.4f\t",matrix[i*dim.first + j]);
		}
		cout << endl;
	}
}